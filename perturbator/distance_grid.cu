#include "hip/hip_runtime.h"
#include "distance_grid.cuh"

#include <algorithm>
#include <cassert>
#include <climits>
#include <cmath>
#include <limits>
#include <stdexcept>
#include <vector>

#include <iostream>

#include <immintrin.h>

namespace doapp {
namespace distance_grid {

__host__ __device__ static std::uint32_t as_u32(float f32) noexcept {
  static_assert(sizeof(float) == sizeof(std::uint32_t));

  std::uint32_t u32;
  memcpy(&u32, &f32, sizeof(float));

  return u32;
}

__host__ __device__ static float as_f32(std::uint32_t u32) noexcept {
  static_assert(sizeof(float) == sizeof(std::uint32_t));

  float f32;
  memcpy(&f32, &u32, sizeof(float));

  return f32;
}

__host__ __device__ static constexpr std::uint32_t
divide_towards_positive_infinity(std::uint32_t numerator,
                                 std::uint32_t denominator) noexcept {
  return numerator / denominator +
         static_cast<std::uint32_t>(numerator % denominator != 0);
}

template <typename T>
__host__ __device__ static constexpr T min(T x, T y) noexcept {
  return (y < x) ? y : x;
}

template <typename T>
__host__ __device__ static constexpr T square(T x) noexcept {
  return x * x;
}

__device__ void atomic_min(volatile float *memory, float value) noexcept {
  float current_f32 = *memory;
  unsigned current = as_u32(current_f32);

  while (!(current_f32 <= value)) {
    current =
        atomicCAS(reinterpret_cast<unsigned *>(const_cast<float *>(memory)),
                  current, as_u32(value));
    current_f32 = as_f32(current);
  }
}

__global__ static void
update_grid(Dimensions dimensions, const float *pointcloud, float *output,
            std::uint32_t num_points,
            std::uint32_t points_per_pointcloud_partition,
            std::uint32_t cells_per_cell_partition,
            std::uint32_t points_to_copy_per_thread,
            std::uint32_t cells_to_write_per_thread) {
  // copy a partition of the pointcloud from global to shared memory

  extern __shared__ float shared[];
  float *result_grid = shared;
  float *pointcloud_partition = result_grid + cells_per_cell_partition;

  // blockIdx.x: pointcloud partition index
  // blockIdx.y: cell partition index

  const float *const global_partition_begin =
      pointcloud + 3 * points_per_pointcloud_partition * blockIdx.x;
  const float *const global_partition_end =
      min(global_partition_begin + 3 * points_per_pointcloud_partition,
          pointcloud + 3 * num_points);

  const float *const this_thread_global_partition_begin =
      global_partition_begin + 3 * points_to_copy_per_thread * threadIdx.x;
  const float *const this_thread_global_partition_end =
      min(this_thread_global_partition_begin + 3 * points_to_copy_per_thread,
          global_partition_end);

  float *this_thread_shared_partition_start =
      pointcloud_partition + 3 * points_to_copy_per_thread * threadIdx.x;
  float *this_thread_shared_partition_end =
      min(this_thread_shared_partition_start + 3 * points_to_copy_per_thread,
          pointcloud_partition + 3 * points_per_pointcloud_partition);

  const float *global_read_head = this_thread_global_partition_begin;
  float *shared_write_head = this_thread_shared_partition_start;

  for (; global_read_head < this_thread_global_partition_end &&
         shared_write_head < this_thread_shared_partition_end;
       global_read_head += 3, shared_write_head += 3) {
    shared_write_head[0] = global_read_head[0];
    shared_write_head[1] = global_read_head[1];
    shared_write_head[2] = global_read_head[2];
  }

  __syncthreads();
  // shared memory is filled in; compute distance grid for a subset of points

  const float x_offset =
      -0.5f * static_cast<float>(dimensions.length) * dimensions.resolution;
  const float y_offset =
      -0.5f * static_cast<float>(dimensions.width) * dimensions.resolution;

  const std::uint32_t cell_offset = cells_per_cell_partition * blockIdx.y;
  const std::uint32_t this_thread_cell_offset =
      cell_offset + cells_to_write_per_thread * threadIdx.x;
  const std::uint32_t num_cells =
      dimensions.length * dimensions.height * dimensions.width;

  for (std::uint32_t cell_index = this_thread_cell_offset;
       cell_index < cell_offset + cells_to_write_per_thread &&
       cell_index < num_cells;
       ++cell_index) {
    const std::uint32_t z_index =
        cell_index / (dimensions.length * dimensions.width);
    const std::uint32_t slice_index =
        cell_index % (dimensions.length * dimensions.width);

    const std::uint32_t y_index = slice_index / dimensions.length;
    const std::uint32_t x_index = slice_index % dimensions.length;

    const float z = static_cast<float>(z_index) * dimensions.resolution;
    const float y =
        static_cast<float>(y_index) * dimensions.resolution + y_offset;
    const float x =
        static_cast<float>(x_index) * dimensions.resolution + x_offset;

    float cell_value_sq = HUGE_VALF;

    for (float *point = this_thread_shared_partition_start;
         point < this_thread_shared_partition_end; point += 3) {
      const float point_x = point[0];
      const float point_y = point[1];
      const float point_z = point[2];

      const float new_distance_sq =
          square(point_x - x) + square(point_y - y) + square(point_z - z);
      cell_value_sq = min(cell_value_sq, new_distance_sq);
    }

    result_grid[cell_index - this_thread_cell_offset] =
        std::sqrt(cell_value_sq);
  }

  const std::uint32_t this_thread_cell_last =
      min(this_thread_cell_offset + cells_to_write_per_thread,
          cells_per_cell_partition);

  volatile float *global_write_head =
      output + cells_per_cell_partition * blockIdx.y;
  const float *shared_read_head = result_grid;

  for (std::uint32_t i = this_thread_cell_offset; i < this_thread_cell_last;
       ++i) {
    atomic_min(&global_write_head[i], shared_read_head[i]);
  }
}

} // namespace distance_grid

DistanceGrid::DistanceGrid(const distance_grid::Dimensions &dimensions) noexcept
    : distances_(dimensions.length * dimensions.width * dimensions.height),
      dimensions_(dimensions) {
  x_offset_ = static_cast<float>(0.5 * static_cast<double>(dimensions_.length) *
                                 static_cast<double>(dimensions_.resolution));
  y_offset_ = static_cast<float>(0.5 * static_cast<double>(dimensions_.width) *
                                 static_cast<double>(dimensions_.resolution));

  x_min_ = -x_offset_;
  x_max_ = x_offset_;

  y_min_ = -y_offset_;
  y_max_ = y_offset_;

  z_max_ = static_cast<float>(static_cast<double>(dimensions_.height) *
                              static_cast<double>(dimensions_.resolution));

  slice_pitch_ = dimensions_.length * dimensions_.height;
}

void DistanceGrid::update(const Matrix<float, Dynamic, 3> &pointcloud) {
  std::fill(distances_.data(), distances_.data() + distances_.size(),
            HUGE_VALF);

  constexpr std::uint32_t SHARED_MEMORY_SIZE =
      48 * (1 << 10); // 48 KiB on Turing architecture

  constexpr std::uint32_t POINTCLOUD_PARTITION_MAX_ALLOCATION =
      SHARED_MEMORY_SIZE / 4;
  constexpr std::uint32_t POINT_SIZE = sizeof(float) * 3; // x y z

  constexpr std::uint32_t MAX_POINTS_PER_POINTCLOUD_PARTITION =
      POINTCLOUD_PARTITION_MAX_ALLOCATION / POINT_SIZE;

  const std::uint32_t num_pointcloud_partitions =
      distance_grid::divide_towards_positive_infinity(
          static_cast<std::uint32_t>(pointcloud.num_rows()),
          MAX_POINTS_PER_POINTCLOUD_PARTITION);
  const std::uint32_t points_per_pointcloud_partition =
      distance_grid::divide_towards_positive_infinity(
          static_cast<std::uint32_t>(pointcloud.num_rows()),
          num_pointcloud_partitions);

  constexpr std::uint32_t GRID_MAX_ALLOCATION =
      SHARED_MEMORY_SIZE - POINTCLOUD_PARTITION_MAX_ALLOCATION;
  constexpr std::uint32_t MAX_CELLS_PER_BLOCK =
      GRID_MAX_ALLOCATION / sizeof(float);

  const std::uint32_t num_cells =
      dimensions_.length * dimensions_.width * dimensions_.height;
  const std::uint32_t num_cell_partitions =
      distance_grid::divide_towards_positive_infinity(num_cells,
                                                      MAX_CELLS_PER_BLOCK);
  const std::uint32_t cells_per_cell_partition =
      distance_grid::divide_towards_positive_infinity(num_cells,
                                                      num_cell_partitions);

  constexpr std::uint32_t BLOCK_SIZE = 1024;

  const std::uint32_t points_to_copy_per_thread =
      distance_grid::divide_towards_positive_infinity(
          points_per_pointcloud_partition, BLOCK_SIZE);
  const std::uint32_t cells_to_write_per_thread =
      distance_grid::divide_towards_positive_infinity(cells_per_cell_partition,
                                                      BLOCK_SIZE);

  distance_grid::
      update_grid<<<dim3(num_pointcloud_partitions, num_cell_partitions, 1),
                    BLOCK_SIZE, SHARED_MEMORY_SIZE>>>(
          dimensions_, pointcloud.data(), distances_.data(),
          static_cast<std::uint32_t>(pointcloud.num_rows()),
          points_per_pointcloud_partition, cells_per_cell_partition,
          points_to_copy_per_thread, cells_to_write_per_thread);

  if (hipDeviceSynchronize() != hipSuccess) {
    throw std::runtime_error("doapp::DistanceGrid::DistanceGrid: update");
  }
}

__host__ __device__ float DistanceGrid::operator()(float x, float y,
                                                   float z) const noexcept {
  if (!(x >= x_min_ && x <= x_max_) || !(y >= y_min_ && y <= y_max_) ||
      !(z >= 0.0 && z <= z_max_)) {
    return HUGE_VALF;
  }

  const auto x_index =
      static_cast<std::uint32_t>((x + x_offset_) / dimensions_.resolution);
  const auto y_index =
      static_cast<std::uint32_t>((y + y_offset_) / dimensions_.resolution);
  const auto z_index = static_cast<std::uint32_t>(z / dimensions_.resolution);

  return distances_[slice_pitch_ * z_index + dimensions_.length * y_index +
                    x_index];
}

} // namespace doapp
