#include "hip/hip_runtime.h"
#include "generate_trajectories.cuh"
#include "common.cuh"
#include <cassert>
#include <cstdio>

__constant__ float initial_waypoint[doapp::num_joints];
__constant__ float final_waypoint[doapp::num_joints];
__constant__ float initial_trajectory[doapp::num_joints * 50];
__device__ unsigned int getTid();
__device__ float getRandFloat(hiprandState* state, float min, float max);
__device__ float getRandFloat(hiprandState* state, float min, float max, float score);

__global__ void init_cudarand(hiprandState* states, unsigned int num_rngs) {
    unsigned int tid = getTid();
    if(tid < num_rngs) {
        //each thread gets the same seed (1234), a different sequence number, and no offset. This should be enough to ensure each RNG is sufficiently random
        hiprand_init(1234, tid, 0, states + tid);
    }
}

__device__ void initalize_trajectories(unsigned int num_waypoints, unsigned int waypoint_dim, hiprandState* rng, float* trajectories) {
    // random trajectories
    if(threadIdx.x < num_waypoints*waypoint_dim) {
            assert(rng);
            float range = doapp::max_joint_angles[threadIdx.x % waypoint_dim] - doapp::min_joint_angles[threadIdx.x % waypoint_dim];
            //min: min - max
            //max: max - min
            trajectories[threadIdx.x] = getRandFloat(rng, -range / 10, range / 10);
    }
}

__device__ void generate_noise_vectors(unsigned int num_noise_vectors, unsigned int noise_vector_dim, float* noise_vectors, hiprandState* rng) {
    if(threadIdx.x < num_noise_vectors*noise_vector_dim) {
        assert(rng);
        noise_vectors[threadIdx.x] = getRandFloat(rng, doapp::min_joint_angles[threadIdx.x % noise_vector_dim], doapp::max_joint_angles[threadIdx.x % noise_vector_dim]);
    }
}

__device__ void compute_noisy_trajectories(unsigned int num_noise_vectors, unsigned int dimensionality, unsigned int num_waypoints, float* noise_vectors, float* trajectories, float* noisy_trajectories) {
    if(threadIdx.x < num_waypoints*dimensionality*num_noise_vectors) {
        int noisy_trajectory_index = threadIdx.x / (num_waypoints*dimensionality);
        noisy_trajectories[threadIdx.x] = noise_vectors[noisy_trajectory_index * dimensionality + (threadIdx.x % dimensionality)] + trajectories[threadIdx.x % (num_waypoints * dimensionality)];
    }
}

__device__ void score_noisy_trajectories(float* noisy_trajectories, unsigned int num_noisy_trajectories, unsigned int num_waypoints, unsigned int waypoint_dim, float* scores, float* accelerations, float* smoothness, volatile int* best_score) {
    //initalize values
    if(threadIdx.x < num_noisy_trajectories) {
        scores[threadIdx.x] = 0;
        smoothness[threadIdx.x] = 0;
    }
    __syncthreads();
    //compute sum of squared accelerations
    if(threadIdx.x < num_noisy_trajectories*num_waypoints*waypoint_dim) {
        unsigned int trajectory = threadIdx.x / (num_waypoints*waypoint_dim);
        atomicAdd(smoothness + trajectory, accelerations[threadIdx.x] * accelerations[threadIdx.x]);
    }
    __syncthreads();
    //multiply by factor of 1/2
    if(threadIdx.x < num_noisy_trajectories) {
        smoothness[threadIdx.x] *= 0.5f;
    }
    __syncthreads();
    //compute cost due to collisions
    float waypoint[doapp::num_joints];
    if(threadIdx.x < num_noisy_trajectories*num_waypoints) {
        //load waypoint. TODO: first transpose noisy_trajectories such that accesses are coalesced. This should result in a decent speedup
        unsigned int trajectory = threadIdx.x / num_waypoints;
        unsigned int waypoint_index = threadIdx.x % num_waypoints;
        for(unsigned int i = 0; i < waypoint_dim; ++i) {
            waypoint[i] = noisy_trajectories[trajectory*num_waypoints*waypoint_dim + waypoint_index*waypoint_dim + i]; //currently, memory accesses between threads are strided by waypoint_dim (AKA not coalesced), and that is not good
        }
        float waypoint_cost = num_collisions(waypoint, doapp::num_joints);
        atomicAdd(scores + trajectory, waypoint_cost); //could do a list reduction, but like, this is one line that does the same thing
    }
    __syncthreads();
    //add smoothness cost
    if(threadIdx.x < num_noisy_trajectories) {
        smoothness[threadIdx.x] *= 0.5f;
        scores[threadIdx.x] += smoothness[threadIdx.x];
        int rounded_score = int(ceilf(scores[threadIdx.x]));
        //printf("thread %d has score %d\n", threadIdx.x, rounded_score);
        atomicMin((int*)best_score, rounded_score);
    }
    //TODO: put in a term about violating joint limits. be sure to move the atomicMin line!
}

__device__ void compute_update_vector(float* scores, float* noise_vectors, unsigned int num_noise_vectors, unsigned int waypoint_dim, float* output, float best_score) {
    if(threadIdx.x < waypoint_dim) {
        output[threadIdx.x] = 0;
    }
    __syncthreads(); //TODO: look into thread groups cause not all threads in the whole block need to sync up here
    if(threadIdx.x < num_noise_vectors*waypoint_dim) {
        unsigned int noise_vector = threadIdx.x / waypoint_dim;
        float our_score = scores[noise_vector];
        float weight = best_score / ceilf(our_score);
        atomicAdd(output + (threadIdx.x % waypoint_dim), weight * noise_vectors[threadIdx.x]);
    }
}

__device__ void update_trajectories(float* trajectory, float* update_vector, unsigned int num_waypoints, unsigned int waypoint_dim) {
    if(threadIdx.x < num_waypoints*waypoint_dim) {
        trajectory[threadIdx.x] -= 10000.0f * update_vector[threadIdx.x % waypoint_dim];
    }
}

__global__ void optimize_trajectories(float* trajectories, float* noise_vectors, float* noisy_trajectories, hiprandState* states, float* velocities, float* accelerations, float* smoothness, float* scores, float* update_vectors, bool* found_better, bool* time_expired, unsigned int num_rngs_per_trajectory, unsigned int num_waypoints, unsigned int waypoint_dim, unsigned int num_noise_vectors, float deltaT) {
    hiprandState* rng = threadIdx.x < num_rngs_per_trajectory ? states + (threadIdx.x + blockIdx.x*num_rngs_per_trajectory) : nullptr;
    float* block_trajectories = trajectories + blockIdx.x*num_waypoints*waypoint_dim;
    float* block_noise_vectors = noise_vectors + blockIdx.x*num_noise_vectors*waypoint_dim;
    float* block_noisy_trajectories = noisy_trajectories + blockIdx.x*num_noise_vectors*num_waypoints*waypoint_dim;
    float* block_velocities = velocities + blockIdx.x*num_noise_vectors*num_waypoints*waypoint_dim;
    float* block_accelerations = accelerations + blockIdx.x*num_noise_vectors*num_waypoints*waypoint_dim;
    float* block_smoothness = smoothness + blockIdx.x*num_noise_vectors;
    float* block_scores = scores + blockIdx.x*(num_noise_vectors + 1) + 1;
    float* block_update_vector = update_vectors + blockIdx.x*waypoint_dim;
    __shared__ volatile int best_score, current_score, best_trajectory_index;
    if(!threadIdx.x) {
        best_score = INT_MAX; //ensure someone sets it
        current_score = INT_MAX;
        best_trajectory_index = -1;
    }

    //TODO: have a shared memory slice of everything read/written to for better access timing
    //initalize_trajectories(num_waypoints, waypoint_dim, rng, block_trajectories);
    //first, score our inital trajectory
    compute_velocity(initial_trajectory, 1, num_waypoints, waypoint_dim, deltaT, block_velocities);
    __syncthreads();
    compute_acceleration(initial_trajectory, 1, num_waypoints, waypoint_dim, deltaT, block_accelerations);
    //TODO: do a list reduction on accelerations (all threads in a block participate) but instead of just adding them, square individual terms first
    __syncthreads();
    //score initial trajectory into block_scores[-1]
    score_noisy_trajectories(initial_trajectory, 1, num_waypoints, waypoint_dim, block_scores - 1, block_accelerations, block_smoothness, &current_score);
    __syncthreads();
    if(!threadIdx.x && !blockIdx.x) {
        printf("inital trajectory score: %d\n", current_score);
    }
    bool use_inital_trajectory = true;
    do {
        //attempt to optimize it
        generate_noise_vectors(num_noise_vectors, waypoint_dim, block_noise_vectors, rng);
        __syncthreads();
        compute_noisy_trajectories(num_noise_vectors, waypoint_dim, num_waypoints, block_noise_vectors, use_inital_trajectory ? initial_trajectory : block_trajectories, block_noisy_trajectories);
        __syncthreads();
        compute_velocity(block_noisy_trajectories, num_noise_vectors, num_waypoints, waypoint_dim, deltaT, block_velocities);
        __syncthreads();
        compute_acceleration(block_velocities, num_noise_vectors, num_waypoints, waypoint_dim, deltaT, block_accelerations);
        //TODO: do a list reduction on accelerations (all threads in a block participate) but instead of just adding them, square individual terms first
        __syncthreads();
        if(!threadIdx.x)
            best_score = INT_MAX; //ensure someone sets it
        __syncthreads();
        score_noisy_trajectories(block_noisy_trajectories, num_noise_vectors, num_waypoints, waypoint_dim, block_scores, block_accelerations, block_smoothness, &best_score);
        __syncthreads();
        //keep the best trajectories
        if(best_score < current_score) {
            if(threadIdx.x < num_noise_vectors*num_waypoints*waypoint_dim) {
                int trajectory = threadIdx.x / (num_waypoints * waypoint_dim);
                if(int(ceilf(scores[trajectory])) == best_score) {
                    block_trajectories[threadIdx.x] = block_noisy_trajectories[threadIdx.x];
                }
            }
            __syncthreads();
            if(!threadIdx.x) {
                best_score = current_score;
                use_inital_trajectory = false;
                printf("New best score for block %d is %d\n", blockIdx.x, best_score);
            }
        } else if(!threadIdx.x) {
            //printf("Block %d did not find a better trajectory\n", blockIdx.x);
        }
        /*
        __syncthreads();
        //TODO: consider also scoring the current trajectory to make sure we don't go in a worse direction
        compute_update_vector(block_scores, block_noise_vectors, num_noise_vectors, waypoint_dim, block_update_vector, best_score);
        __syncthreads();
        update_trajectories(block_trajectories, block_update_vector, num_waypoints, waypoint_dim);
        */
    //} while(++count < 5);
    } while(!*time_expired);
    if(!threadIdx.x) {
        printf("Found better: %s\n", use_inital_trajectory ? "false" : "true");
        found_better[blockIdx.x] = !use_inital_trajectory;
    }
}

__device__ void compute_acceleration(float* velocity, unsigned int num_noisy_trajectories, unsigned int num_waypoints, unsigned waypoint_dim, float deltaT, float* output) {
    compute_derivative(velocity, num_noisy_trajectories, num_waypoints, waypoint_dim, deltaT, output);
}
__device__ void compute_velocity(float* input_trajectories, unsigned int num_noisy_trajectories, unsigned int num_waypoints, unsigned waypoint_dim, float deltaT, float* output) {
    compute_derivative(input_trajectories, num_noisy_trajectories, num_waypoints, waypoint_dim, deltaT, output);
}
__device__ void compute_derivative(float* input, unsigned int num_noisy_trajectories, unsigned int num_waypoints, unsigned int waypoint_dim, float deltaT, float* output) {
    if(threadIdx.x < num_noisy_trajectories*num_waypoints*waypoint_dim) {
        unsigned int waypoint = (threadIdx.x / waypoint_dim) % num_waypoints;
        unsigned int dim = threadIdx.x % waypoint_dim;
        float prior_val, current_val;
        prior_val = waypoint ? input[threadIdx.x - waypoint_dim] : initial_waypoint[dim];
        current_val = input[threadIdx.x];
        float val = (current_val - prior_val) / deltaT;
        output[threadIdx.x] = val;
    }
}

__device__ unsigned int getTid() {
    return threadIdx.x + blockIdx.x*blockDim.x;
}

/*
__device__ float getRandFloat(hiprandState* state, float min, float max) {
    return hiprand_normal(state) * (max - min + 0.999999) + min;
}
*/

__device__ float getRandFloat(hiprandState* state, float min_val, float max_val) {
    return hiprand_normal(state);
    //return min(max(hiprand_normal(state) * 0.01f, min_val), max_val);
}
__device__ unsigned int num_collisions(float* waypoint, unsigned int waypoint_dim) {
    return 0; //TODO: use Greg's impl. For testing, might want to see if we can inverse the order of the trajectories based upon this or something idk
}
