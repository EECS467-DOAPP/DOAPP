#include "hip/hip_runtime.h"
#include "generate_trajectories.cuh"
#include <cfloat>
#include <cassert>

__device__ unsigned int getTid();
__device__ float getRandFloat(hiprandState& state);

__global__ void init_cudarand(hiprandState* states, unsigned int num_rngs) {
    unsigned int tid = getTid();
    if(tid < num_rngs) {
        //each thread gets the same seed (1234), a different sequence number, and no offset. This should be enough to ensure each RNG is sufficiently random
        hiprand_init(1234, tid, 0, states + tid);
    }
}

__device__ void initalize_trajectories(unsigned int num_waypoints, unsigned int waypoint_dim, hiprandState* rng, float* trajectories) {
    if(threadIdx.x < num_waypoints*waypoint_dim) {
            assert(rng);
            trajectories[threadIdx.x] = getRandFloat(*rng);
    }
}

__device__ void generate_noise_vectors(unsigned int num_noise_vectors, unsigned int noise_vector_dim, float* noise_vectors, hiprandState* rng) {
    if(threadIdx.x < num_noise_vectors*noise_vector_dim) {
        assert(rng);
        noise_vectors[threadIdx.x] = getRandFloat(*rng);
    }
}

__device__ void compute_noisy_trajectories(unsigned int num_noise_vectors, unsigned int dimensionality, unsigned int num_waypoints, float* noise_vectors, float* trajectories, float* noisy_trajectories) {
    if(threadIdx.x < num_waypoints*dimensionality*num_noise_vectors) {
        int noisy_trajectory_index = threadIdx.x / (num_waypoints*dimensionality);
        noisy_trajectories[threadIdx.x] = noise_vectors[noisy_trajectory_index * dimensionality + (threadIdx.x % dimensionality)] + trajectories[threadIdx.x % (num_waypoints * dimensionality)];
    }
}

__global__ void optimize_trajectories(float* trajectories, float* noise_vectors, float* noisy_trajectories, hiprandState* states, unsigned int num_rngs_per_trajectory, unsigned int num_waypoints, unsigned int waypoint_dim, unsigned int num_noise_vectors) {
    hiprandState* rng = threadIdx.x < num_rngs_per_trajectory ? states + (threadIdx.x + blockIdx.x*num_rngs_per_trajectory) : nullptr;
    //TODO: have a shared memory slice of everything read/written to for better access timing
    initalize_trajectories(num_waypoints, waypoint_dim, rng, trajectories);
    generate_noise_vectors(num_noise_vectors, waypoint_dim, noise_vectors, rng);
    compute_noisy_trajectories(num_noise_vectors, waypoint_dim, num_waypoints, noise_vectors, trajectories, noisy_trajectories);
}

__device__ unsigned int getTid() {
    return threadIdx.x + blockIdx.x*blockDim.x;
}

__device__ float getRandFloat(hiprandState& state) {
    return hiprand_uniform(&state) * FLT_MAX;
}
